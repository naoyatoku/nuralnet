#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <random>
#include <iostream>
#include <vector>
using namespace std;
/*
__global__ void matrix_2d_kernel(float* w , float* M , float* N ) {
    //blockは一個(blockIdx=0)
    //これで1blockあたりx:m y:n 個のスレッドがある 
    // row_idx  = threadIdx.x 
    // idx = threadIdx.y

    atomicAdd( N + threadIdx.y , w[threadIdx.x + ( threadIdx.y*blockDim.x) ] * M[threadIdx.x] );
    printf("x[%d]y[%d] : w[%d][%d]=%f  M[%d]:%f  -> N[%d]:%f \n"
        , threadIdx.x, threadIdx.y
        , threadIdx.y, threadIdx.x, w[threadIdx.x + (threadIdx.y * blockDim.x)]
        , threadIdx.x, M[threadIdx.x]
        , threadIdx.y, N[threadIdx.y]
    );
// 
//    N[0] = 0.1;
}


__host__ void initialize(float* d, int size)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 1.0);

    // 配列wの初期化
    for (int i = 0; i < size; ++i) {d[i] = dis(gen);}

    // 結果を表示（オプション）
    for (int i = 0; i < size; ++i) { printf("%d[%lf]\n", i, d[i]); }
}


*/


//重みづけマトリクスは別に?
//レイヤー間で、
//  (前段ノードの数)
//各レイヤーごとのノード数マトリクスです。これは、netを作るときに数が行列の数が決まるので
//float *w[MAX_LAYER];   //これがレイヤーの間にある。ー＞レイヤーの数分あるこれらは全部動的に作ってみる。
//継承を使わないノードを作る
struct node
{
    float   a;      //入力
    float   da_de;  //
    float   y;      //


    __device__  float activate(){       //relu限定にしてみる。
        if(a<0)return 0.0;
        return a;
    }
    __device__  float d_activate()
    {
        if(a>0.0){  return 1.0;}
        return 0.0;       //
    }
    //
};
template<size_t _n>
struct layer
{
    int n;
    node nodes[_n];
    layer():n(_n){;}
};

//継承しないnet
template<size_t _n>
struct net
{
    layer*layers;
    int n_layer;
    net(int l):n_layer(l){;}
};

//なんか
void do_net()
{
    int layers=0;       //これがnetかな？
    //構成するネットを作ります。
    int nodes[] = {16,8,36,-1};     //とりあえず固定で
    const   int layers = 3;         //３レイヤー

//    net _net<3>;
}





 
int main() {

/*
    //n次元ベクトル→m次元ベクトルへ変換する線形写像行列計算、

    const int m = 4;       //今段レイヤー数(ベクトル次元)
    const int n = 5;        //次段レイヤー数(ベクトル次元)

    //  ベクトルを縦にすると、
    // ベクトルは（m*n）
    // 
    //  行列A() * ベクトルm => ベクトル n
    //

    //重みづけを定義します。
    float w[n][m]={ {1,0,0,0},{0,1,0,0},{0,0,1,0},{0,0,0,1},{1,0,0,1}};         //m列がn行で縦のベクトル[n]ができる
    float M[m] = { 1,2,3,4 }, N[n] = {0,0,0,0,0};              //各ベクトルです。
    //0-1の間で初期化
    initialize(&w[0][0],n*m);   //
//    initialize(&M[0],m);
//    initialize(&N[0],n);

    float* w_dev;            //デバイス側のメモリポインタ
    float* M_dev,*N_dev;      //
    
    // デバイスメモリを確保
    {
        hipMalloc((void**)&w_dev, sizeof(w));
        hipMalloc((void**)&M_dev, sizeof(M));
        hipMalloc((void**)&N_dev, sizeof(N));
        hipMemcpy( w_dev , &w[0][0]    , sizeof(w), hipMemcpyHostToDevice);
        hipMemcpy( M_dev , &M[0]       , sizeof(M), hipMemcpyHostToDevice);
        hipMemcpy( N_dev , &N[0]       , sizeof(N), hipMemcpyHostToDevice);
    }

    // カーネルを起動
//    kernel<<blocks_per_grid,threads_per_block>>()
//    dim3 threads_per_block(n,m);
   // dim3 blocks(1, 0);
 //  kernel<<< 1 ,threads_per_block >>>(w_dev,M_dev,N_dev);
//    kernel<<< 1 ,256 >>>(w_dev,M_dev,N_dev);

    dim3 threads_per_block(m, n);
    matrix_2d_kernel << <1, threads_per_block >> > (w_dev, M_dev, N_dev);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize();


    // 結果をホストにコピー
    hipMemcpy( (void*)&N[0] , N_dev,  sizeof(N), hipMemcpyDeviceToHost);


    for (int i = 0; i < n; ++i) {
        printf("%f\n" , N[i]);
    }
//    printf("Result: %d\n", hostData[0]);

    // デバイスメモリを解放
    hipFree(w_dev);
    hipFree(M_dev);
    hipFree(N_dev);
*/
    return 0;
}

//
//    N[0]   = M[0]*w[0][0] + M[1]* w[0][1] + ....  M[m]*w[0][m-1]
  //  N[1]   = M[0]*w[1][0] ....................... M[m]*w[1][m-1]
  //                             :
  //  N[n-1] = M[0]*w[n-1][0] + ..................  M[m]*w[n-1][m-1]


    //行列でやってみる。
    //ニューラルネットの計算は、重みづけが
    //横が次レイヤーノード数 n
    //縦が今レイヤーノード数 m
#if 0
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

//-----------------------------------------------------------------------------
// backwardの計算
//-----------------------------------------------------------------------------


損失レイヤーのdE/da計算
virtual double activate(void) {
    //ここで本来重みづけは、w[0,0,1,0,0,0,]というようなn番目が1の配列だが、結果n番目の入力そのものなのでそれを採用するようにします。
//		_a		=	in->at(_n)->out();		//前段の出力をそのまま使います。
        //損失関数を呼びます。
    _Assert(_in != nullptr, "error_perceptron::activate() : ");      //_この時点でinが確定している必要があります。(affine呼び出しで登録される。)
    _out = _loss->E(_a, _t, this);
    損失関数で出力を決める。

    //ここでこのオブジェクトの勾配も求まります。
    _dE_da = _loss->dE_dy(_a, _t);

    //
    dE_dy()は損失関数の結果です。

    return _out;
}


//損失レイヤーでは損失関数によりdE/daが計算されている。


//損失レイヤーの一つ手前から始める。
for (int l = 損失レイヤの一つ手前; 最初のレイヤーまで; l--)
{
    kernel
    for (lのノードサイズ) {
        double de_dy = 0; //今回の更新
        //kernel
        for (l + 1のノードサイズ) {
            de_dy[i]  +=  net[l + 1]->node[j]->dE_da() * net[l + 1]->node[j]->_w[i];        //Σの部分
                    ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^      ^^^^^^^^^^^^^^^^^^^^^^^^^^^^
            de_dy[i]   =  Σ      次段ノード[j] のdE / da           ×      次段のノード[j] の  w[i]
            ^^^^^
           ※ ノード[i]のde_dy(出力結果の損失度合)
        }

        net[l]->node[i]->set_dE_da(de_dy* net[l]->node[i]->d_act());        //σの部分
        ※ノード[i]のdE_da =  de_dy[i] * d_act[i]アクティベータの微分


        ｗ更新_wのサイズは前段ノードの数です。
        //
        for (int x = 0; x < net[l]->node[i]->_w.size(); ++x) {      //
            net[l]->node[i]->_w[x] -= _lr * net[l]->node[i]->dE_da() * net[l]->node[i]->in(x);
                //^^^^^^^^^^^^^^^^^^^^ da[L]i/dw[L]xi => a[L]iへの前段[L-1]のnode[x]の出力	( = net[l]->node[i]->in(x) )
                ※ w[x]  -= （学習率） *  dE/da[i] * 入力値(i)<- (x)
         }
    }

}
//各ノード
//ベクトルと、次段レイヤーの全ノードに対する





void backward(vector<layer*>& net) {        //layer：forwardを行ったネット。vector<double>tは、
    //layersには、入力層[0]+中間層[1]-[N]+損失関数のあるレイヤー[N+1]でなっている。更新する必要があるのは、[1]-[N]の層のw。
    //layaer.size()は、損失関数のレイヤも含む. 更新する必要がある層の添え字は、[size()-2]->[1]
    for (int l = net.size() - 2; l > 0; --l) {       //レイヤーの数分です。（入力レイヤーは除く）
        for (int i = 0; i < net[l]->node.size(); ++i) {                 //1つのレイヤーの各ノードに対して最適化していきます。
            //まずdE/da(L)[i]を作ります。一番最後のレイヤーから実行する前提で書いています。（はじめからやるとうまくいきません。)
            {
                double de_dy = 0; //今回の更新
                for (int j = 0; j < net[l + 1]->node.size(); ++j) {
                    de_dy += net[l + 1]->node[j]->dE_da() * net[l + 1]->node[j]->_w[i];        //Σの部分
                }
                net[l]->node[i]->set_dE_da(de_dy * net[l]->node[i]->d_act());        //σの部分
            }
            //dE/da(L)iが決まったので、wを更新していきます。
            for (int x = 0; x < net[l]->node[i]->_w.size(); ++x) {      //
                net[l]->node[i]->_w[x] -= _lr * net[l]->node[i]->dE_da() * net[l]->node[i]->in(x);
                //^^^^^^^^^^^^^^^^^^^^ da[L]i/dw[L]xi => a[L]iへの前段[L-1]のnode[x]の出力	( = net[l]->node[i]->in(x) )
            }
            //バイアスを更新です。
            net[l]->node[i]->_b -= _lr * net[l]->node[i]->dE_da();
        }
    }
}


#endif
