//�o�b�t�@����
//�G�X�P�[�v�V�[�P���X�p�ł��B
#include <stdarg.h>
#include "common.h"
#include "string.h"

char _esc_seq_buf[1024];


__host__ __device__ void pos_printf(int x, int y, char* fmt,...) {
    char buf[256];
    sprintf_s(buf,sizeof(buf), "\x1B[%d;%dH%s", y, x,fmt);
    va_list ap;    va_start(ap, fmt);
    vprintf_s(buf, ap);
//    printf("\x1B[%d;%dH", y, x );
}
__host__ __device__ void esc_clr()
{
    printf("\x1B[2J");
}
