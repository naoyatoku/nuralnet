//�o�b�t�@����
//�G�X�P�[�v�V�[�P���X�p�ł��B
#include <stdarg.h>
#include "common.cuh"
#include "string.h"
#include <iostream>

char _esc_seq_buf[1024];

//
__host__  void pos_printf(int x, int y, char* fmt,...) {
    char buf[256];
    sprintf_s(buf,sizeof(buf), "\x1B[%d;%dH%s", y, x,fmt);
    va_list ap;    va_start(ap, fmt);
    vprintf_s(buf, ap);
//    printf("\x1B[%d;%dH", y, x );
}
__host__ __device__ void esc_clr()
{
    printf("\x1B[2J");
}
__device__ void _pos(int x, int y) {
    printf("\x1B[%d;%dH", y, x);
}

__host__
void dumpGPUInfo()
{

    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    std::cout << "Number of SMs: " << numSMs << std::endl;

    {
        int device;
        hipDeviceProp_t prop;

        // �f�o�C�X�̎擾
        hipGetDevice(&device);

        // �f�o�C�X�v���p�e�B�̎擾
        hipGetDeviceProperties(&prop, device);
        printf("CUDA Capability Major/Minor version number: %d.%d\n", prop.major, prop.minor);
    }

}