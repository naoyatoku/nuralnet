#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "common.cuh"
#include "node.cuh"




//-----------------------------------------------------------------------------------------------------------------
// netを使用するnodeメンバ関数をここに
//-----------------------------------------------------------------------------------------------------------------
//これカーネル。
__global__ void _calc_max_kernel(const node* nd, int n_node,float *_max)
{
    int x = threadIdx.x;                //本スレッドが担当するノード
    //一番近い2のs累乗の数字を探す。
    __shared__ float max[256];          //※このメモリはカーネルが終わるまでなくなりません。また、固定のサイズでしか取れません。
    __shared__ int _n;                  //最大値を求める際のリダクションの数（2の累乗の数：でないとリダクションが成立しない）
    //リダクションの数を決めます.n_node に一番近い
    if (x == 0) {
        _n = 1;
        while (_n < n_node) {   //2倍していき、n_ndに一番近いを探します。
           _n <<= 1;
        }
        _Assert(sizeof(max) / sizeof(float) > _n, "max size is not enough");
    }
    __syncthreads();
    //ここで_nが定まりました。
    // max初期化します。
    if (x < _n) {
        max[x] = (x < n_node) ? nd[x].a : -10000000000.0;
    }
    __syncthreads();
    //リダクションしていきます。
    for (int n = _n / 2; n > 0; n /= 2) {                                  //N=7 : n=3         n=1
        if (x < n) {                                                        //thread.xは0,1,2 
            if (max[x] < max[x + n]) {
                max[x] = max[x + n];
            }
        }
        __syncthreads();
    }
    //ここでmax[0]に最大値はいっているはず
#if 1  //debug（検算です）
    {
        if (x == 0) {
            float m = -100.0;
            for (int i = 0; i < n_node; ++i) {
                if (nd[i].a > m) {
                    m = nd[i].a;
                }
            }
            _Assert(max[0] == m, "max different");
        }

    }
#endif
    __syncthreads();

    *_max  = max[0];     //これが小舘です
    return;

}
//これがカーネルを呼び出します。呼び出したデバイスの状態に関係なく必要なスレッドを呼びたいため。
__device__ float calc_max( const node *nd , int size)
{
    _Assert(size < 512, "calcmax() thread num overflw");    //一番近い2のべき乗のスレッド数が必要なためスレッドｘの最大値は1024なので512を最大とします。これより大きなものの比較はまた別に考えます。
    float max;
    dim3 threads(size*2, 1, 1);                          //もしバッチでやりたい場合はブロック数またはyを増やすか
    dim3 blocks(1, 1, 1);
    _calc_max_kernel << < blocks, threads >> > (nd,size,&max);   //
    return max;
}
