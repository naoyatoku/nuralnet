#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include "common.h"

//#include "node.h"
#include "net.h"

void dumpGPUInfo();

#define     NET_N_LAYER     3

//入力データ、回答データもGPUへ送り、GPU内でループさせたい。
//学習が終わるまで

__global__ void fwd_kernel(net<NET_N_LAYER>* _net)
{
    int cur_idx    = threadIdx.y;          //値を求めるノードの番号です。ブロックに割り付けます。
    int prev_idx   = threadIdx.x;         //全結合を行う全段のノードです。
    //このスレッドが、次レイヤーのどのインデックスのノードに対し担当するか：block_idx
    for (int l = 1; l < _net->n_layers; ++l) {
        layer& L_pre = _net->layers[l - 1], L_cur = _net->layers[l];
        node& prev = L_pre.nodes.gpu[prev_idx];
        node& cur = L_cur.nodes.gpu[cur_idx];

        if ( cur_idx < L_cur.nodes.size && prev_idx < L_pre.nodes.size) {


            // 前段ノードすべてから、今回のノード cur_node_idxのノードへ全結合が行われる。
            atomicAdd(&cur.a, prev.y * cur.w.gpu[prev_idx]);
        }

        __syncthreads();  // 全スレッドが同期する場所は、ループ内の条件外に配置する

        //=============================================================================
        //  activate
        //=============================================================================
        if (prev_idx == 0 && cur_idx < L_cur.nodes.size) {      //  全結合が終わったので
            node& cur = L_cur.nodes.gpu[cur_idx];
            cur.a += cur.b;
            cur.y = cur.activate();
        }
        __syncthreads();  // さらに必要なら、もう一度同期を挟む
    }
    //ここで、
    //最終段のdE/daを求めます。
    _net->loss_softmax_with_crossentropy();     //
    //
}

int main() {

    dumpGPUInfo();

    net<NET_N_LAYER> myNet(3,5,2);  // 3ノード、5ノード、2ノードを持つネットワークを作成
    //net自身をgpuへコピーします。
    //input作る
    for (int i = 0; i < myNet.layers[0].nodes.size; ++i) {
        myNet.layers[0].nodes.cpu[i].y = .5 * (1+i);
    }

    net<NET_N_LAYER>* pnet = myNet.Transfer_to_GPU();        //netの中身を転送します。

    dim3 threads(16, 16, 1);                             //スレッドとしてパーセプトロンを動作させるようにします。これがバッチ数分ある。
    dim3 blocks(1, 1, 1);                              //とりあえずバッチ分をこちらにこれが128個あるはず
    fwd_kernel << < blocks, threads >> > (pnet);

    hipDeviceSynchronize();                            //


    //GPU→CPUへ計算結果を転送する。
    myNet.Transfer_to_CPU();                            //
    myNet.dump();


//    if ((err = hipGetLastError())!= hipSuccess)    {      printf("Error: %s\n", hipGetErrorString(err));         return;    }
    return 0;
}

void dumpGPUInfo()
{

    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    std::cout << "Number of SMs: " << numSMs << std::endl;

}

