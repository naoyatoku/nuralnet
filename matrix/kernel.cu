#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include "common.cuh"
#include "net.cuh"

#define     NET_N_LAYER     3

//入力データ、回答データもGPUへ送り、GPU内でループさせたい。
//学習が終わるまで

__global__ void fwd_kernel(net* _net)
{
    int cur_idx    = threadIdx.x;          //値を求めるノードの番号です。ブロックに割り付けます。
    int prev_idx   = threadIdx.y;         //全結合を行う前段のノードです。
    //このスレッドが、次レイヤーのどのインデックスのノードに対し担当するか：block_idx
    for (int l = 1; l < _net->ly.size; ++l) {
        layer& L_pre = _net->ly(GPU,l - 1), L_cur = _net->ly(GPU,l);
        node& prev = L_pre.nodes.gpu[prev_idx];
        node& cur = L_cur.nodes.gpu[cur_idx];
        //============================================================================
        //   affine
        //============================================================================
        if ( cur_idx < L_cur.nodes.size && prev_idx < L_pre.nodes.size) {
            // 前段ノードすべてから、今回のノード cur_node_idxのノードへ全結合が行われる。
            atomicAdd(&cur.a, prev.y * cur.w.gpu[prev_idx]);
            if(prev_idx==0){        //対象となるノードんに対して一つのスレッドだけが行います。
                cur.a += cur.b;     //bias
            }
        }

        __syncthreads();  // 全スレッドが同期する場所は、ループ内の条件外に配置する

        //=============================================================================
        //  activate
        //=============================================================================
        //ここはthreadIdx.xのみ、かつ同レイヤーのノード数しか入ってこない。そのため、内部でmaxの計算を擦る際に、ノード数以上のスレッドが必要になってしまう
        if (prev_idx == 0 && cur_idx < L_cur.nodes.size) {      //  全結合が終わったので  : prev_idx==0 はthread.x , 各ノードは threadIdx.y
            //これからの処理は対象レイヤーの各ノード分のスレッドのみ
            node& cur = L_cur.nodes.gpu[cur_idx];
            //
#if 1
            if(l==2)_net->dump(GPU);
#endif
            cur.y = cur.activate(&L_cur);     //ノードのactivateでsoftmaxの場合、全部のmaxとsumが必要
        }
        __syncthreads();  // さらに必要なら、もう一度同期を挟む
    }
#if 1
    if (threadIdx.y < _net->ly.size ) {
        if (threadIdx.x < _net->ly(GPU,threadIdx.y).nodes.size) {
            int l = threadIdx.y; int n = threadIdx.x;
            printf("[%d][%d]a;%fy;%f\n", l , n , _net->ly(GPU,l).nodes(GPU,n).a, _net->ly(GPU,l).nodes(GPU,n).y );
        }
    }
    __syncthreads();  // さらに必要なら、もう一度同期を挟む

#endif
    //ここで、
    //最終段のdE/daを求めます。
//    _net->loss_softmax_with_crossentropy();       //
//    _net->loss_softmax_with_crossentropy();
//    _net->test();
//    _net->loss_softmax_with_crossentropy();
}

