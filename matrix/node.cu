#include "hip/hip_runtime.h"
﻿#include "common.h"
#include <random>
#include "node.h"

using namespace std;
#include<stdio.h>

//乱数を発生させるためのシードです。
static random_device rd;
static mt19937 gen(rd());
//static std::normal_distribution<float> distribution(0.0,1.0);		//標準偏差で初期化する。

//    cpu_gpu_adr<weight> __w;             //重みづけメモリのポインタを持ってみます。これはGPUだったり、CPUだったりする。

//Xavier初期化
__host__ void node::__w_init_Xavier(void){
	_Assert(  w.size > 0, "w_init_xavier() : in size is 0");
//	std::default_random_engine generator;
	std::normal_distribution<double> distribution(0.0, std::sqrt(1.0 / w.size ));
    for(int i=0 ; i < w.size ; ++i ){   w.cpu[i] = distribution(gen); }
}
//He初期化
__host__ void node::__w_init_He(void){
//	int fan_in = _w.size();					//前段のノードの数は、_wのサイズに割り当てられています。
//	std::default_random_engine generator;
	_Assert(w.size > 0, "w_init_He() : in size is 0");
	std::normal_distribution<double> distribution(0.0, std::sqrt(2.0 / w.size ));
    for(int i=0 ; i < w.size ; ++i ){  
		w.cpu[i] = distribution(gen);
	}
}
//正規分布による初期化
__host__ void node::__w_init_std(void){
	_Assert( w.size > 0, "w_init_std() : in size is 0");
	normal_distribution<> dist(0.0,.01);            //
    for(int i=0 ; i < w.size ; ++i ){
		w.cpu[i] = dist(gen);        
		printf("%lf,", w.cpu[i]);
	}
	printf("\r\n");
}
