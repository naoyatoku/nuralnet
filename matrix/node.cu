#include "hip/hip_runtime.h"
﻿#include "common.cuh"
#include <random>
#include "node.cuh"

using namespace std;
#include<stdio.h>

//乱数を発生させるためのシードです。
static random_device rd;
static mt19937 gen(rd());
//static std::normal_distribution<float> distribution(0.0,1.0);		//標準偏差で初期化する。

//    cpu_gpu_adr<weight> __w;             //重みづけメモリのポインタを持ってみます。これはGPUだったり、CPUだったりする。

//Xavier初期化
__host__ void node::__w_init_Xavier(void){
	_Assert(  w.size > 0, "w_init_xavier() : in size is 0");
//	std::default_random_engine generator;
	std::normal_distribution<double> distribution(0.0, std::sqrt(1.0 / w.size ));
    for(int i=0 ; i < w.size ; ++i ){   w.cpu[i] = distribution(gen); }
}
//He初期化
__host__ void node::__w_init_He(void){
//	int fan_in = _w.size();					//前段のノードの数は、_wのサイズに割り当てられています。
//	std::default_random_engine generator;
	_Assert(w.size > 0, "w_init_He() : in size is 0");
	std::normal_distribution<double> distribution(0.0, std::sqrt(2.0 / w.size ));
    for(int i=0 ; i < w.size ; ++i ){  
		w.cpu[i] = distribution(gen);
	}
}
//正規分布による初期化
__host__ void node::__w_init_std(void){
	_Assert( w.size > 0, "w_init_std() : in size is 0");
	normal_distribution<> dist(0.0,.01);            //
    for(int i=0 ; i < w.size ; ++i ){
		w.cpu[i] = dist(gen);        
		printf("%lf,", w.cpu[i]);
	}
	printf("\r\n");
}
__device__ float node::activate(const layer *parent)
{  // ReLU限定にしてみる
    switch(act_type){
    case ACT_SOFTMAX:
        return activate_softmax(parent);
        break;
     case  ACT_RELU:
         return (a < 0.0f) ? 0.0f : a;
     case ACT_NOP:
         return a;
         break;
    }
}

__device__ float node::d_activate()
{
    switch (act_type) {
    case ACT_SOFTMAX:
        return d_activate_softmax();
        break;
    case ACT_RELU:
        return (a > 0.0f) ? 1.0f : 0.0f;
    case ACT_NOP:
    default:
        return 1.0f;
        break;
    }
}
__host__ void node::alloc_w(int n_prev_nodes)
{
    w.alloc(n_prev_nodes);
    __w_init_std();             //適当な値で初期化します。
}
//内包メンバのGPU転送です。
__host__ void node::Transfer_contained_members_to_GPU() 
{
    w.Transfer_to_GPU();
}
__host__ void node::Transfer_contained_members_to_CPU() {
    w.Transfer_to_CPU();    
}
__host__ __device__
void node::dump(int l,int n)const 
{
    _pos(l * 40, n + 1);
    printf(  "a[%8.6f]y[%8.6f]dEda[%8.6f]",a,y,dE_da );
}

//ノードののアクティベーションはmaxを計算するためにnetに依存する。が、すごくいやだ。

#include "layer.cuh"    //ここでレイヤーの情報を得るのはいいらしい。
__device__ float node::activate_softmax(const layer*p_parent_layer)
{
//    const layer& _l = _net->layers[this->nlyr];        //自分が所属するレイヤーです。
//    float max;
    if (threadIdx.x == 0) {
        p_parent_layer->request(REQ_CALC_MAX);

//        parent.request(0);
//            max_kernel <<< 16, 1 >>> (0);
    }
//    __syncthreads();
    return 0.0;
}
__device__ float node::d_activate_softmax(){            //
    return 0.0;
}
