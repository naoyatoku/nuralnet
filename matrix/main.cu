﻿
#include "hip/hip_runtime.h"
#include <iostream>

#include "net.cuh"
#include "common.cuh"
#include "kernels.cuh"

int main() {

    dumpGPUInfo();

    //netをセットアップする。面倒か。。。？
    cpu_gpu_mem<net> _net(1);
    _net().construct(3, 3, 5, ACT_RELU, 7, ACT_SOFTMAX, LOSS_SOFTMAX_WITH_CROSSENTROPY);
    //入力を作らないと、とりあえず適当な数値です。
    for (int i = 0; i < _net().ly(CPU, 0).nodes.size; ++i) {
        _net().ly(CPU, 0).nodes(CPU, i).y = (i * 1) * 0.5;
    }



    _net.cpu->dump();
    //GPUへの転送作業です。
    _net().Transfer_contained_members_to_GPU();
    _net.Transfer_to_GPU();


    dim3 threads(16, 16, 1);                             //スレッドとしてパーセプトロンを動作させるようにします。これがバッチ数分ある。
    dim3 blocks(1, 1, 1);                              //とりあえずバッチ分をこちらにこれが128個あるはず
    fwd_kernel << < blocks, threads >> > (_net.gpu);

    hipDeviceSynchronize();                            //


    //GPU→CPUへ計算結果を転送する。
//    myNet.Transfer_to_CPU();                            //
//    myNet.dump();


//    if ((err = hipGetLastError())!= hipSuccess)    {      printf("Error: %s\n", hipGetErrorString(err));         return;    }
    return 0;
}


