#include "hip/hip_runtime.h"
#include "net.cuh"

#include <stdarg.h>

//----------------------------------------------------------------------
//      constructor
//----------------------------------------------------------------------
 __host__ void  net::construct(int n_layers , int n_input ,int n_mid_nodes , int act_type , int n_output_nodes, int output_act_type , int loss_type)
{
    //レイヤーの割付です。
    ly.alloc(n_layers);
    //各レイヤーのノードを割り付けていきます。
    {
        ly(CPU,0).alloc_nodes(0,n_input);                           //1番目
        for (int i = 1; i < ly.size-1; ++i ) {                        //隠れそう
            ly(CPU,i).alloc_nodes(i,n_mid_nodes , act_type, &ly(CPU,i-1) );
        }
        ly(CPU, n_layers - 1).alloc_nodes(n_layers - 1, n_output_nodes, output_act_type , &ly(CPU,n_layers -2 ) );       //出力層
    }
}
//----------------------------------------------------------------------
//      CPU--->GPU転送
//----------------------------------------------------------------------
__host__ void net::Transfer_contained_members_to_GPU()
{
    //nodeの転送です。
    for (int l = 0; l < ly.size; ++l) {
        ly(CPU,l).Transfer_contained_members_to_GPU();
    }
    //layerの転送です。
    ly.Transfer_to_GPU();
}
//----------------------------------------------------------------------
//      CPU<---GPU  転送
//----------------------------------------------------------------------
__host__ void net::Transfer_contained_members_to_CPU() 
{
    //net自体を転送します。
    for (int l = 0; l < ly.size ; ++l) {
        ly(CPU,l).Transfer_contained_members_to_CPU();
    }
    //net自体は、GPU上でのものと変わりはないのでコピーしない。（GPUの先で必要になったらコピーする。）
}
__host__ __device__ 
void net::dump(int locate)const
{
    esc_clr();
    for (int l = 0; l < ly.size; ++l) {
        ly(locate,l).dump(l,locate);
//            printf("\r\n");
    }
}

//入力と、正解の組み合わせをセットして、学習を行えるようにする。
__host__ void net::set_input_answer(float *in,int in_size , float *answer,int ans_size)
{
    //入力は、入力段コピーするようにします。
    _Assert(in_size ==  ly(CPU,0).nodes.size , "net::set_input_answer : in_size illegale(%f,%f)" , in_size , ly(CPU,0).nodes.size );
    for(int i=0 ; i < in_size ; ++i){
        ly(CPU, 0).nodes(CPU, i).a = in[i];
    }
}