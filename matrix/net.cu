#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "net.h"

/*
//gpu用に損失関数と、その各ノードのdE/daを求めるように
__device__ loss_E_softmax_with_crossentropy(float y, float t , float &softmax_y)
{
    float sum = 0.0;
    {
            float max = -100000.0;	//マイナスの大きな値にしておきます。
            //まずmaxを計算します。
            //
            //tokuこれがだめ。pは、自分のノード。
            //同じ列の入力を見ないといけないので、in_array()のout()を見る必要があります。
            //affineの後なので、本来、同レイヤーの
            for (int i = 0; i < in_size() ; ++i) {
                if ( in_array[i].out() > max) { max = in_array[i].out(); }
            }	//maxを算出していきます。
            for (int i = 0; i < in_size() ; ++i) {
                sum += exp( in_array[i].out() - max);
//                if (_idx == 0) {
//                    printf("  softmax: in[%d] : %lf \n" , i , in_array[i].out());
//                }
            }	//これはaffin後の入力です。


            //今回の活性化の値は、以下です。
            //		_Assert(sum!=0.0 , "softmax::act() : sum is zero");
            if (sum == 0.0)sum = 0.00000001;	//0だった場合にはとても小さな値にしてみる。
            softmax_y = exp( a() - max) / sum;//ソフトマックスと、クロスエントロピー誤差を組み合わせた損失関数
//            printf("softmax[%d]=%lf   max=%lf , sum=%lf , in.out:%lf  a:%lf\n " , _idx , softmax_y , max ,sum , in_array[_idx].out() , _a);
            //^^^^^^
    }
}
__device__ float loss_dE_dy_softmax_with_crossentropy(double y, double t)
{
        //クロスエントロピーと、ソフトマックスが全結合なし（一対一でつながっている）とき、
        //  
        //　この層の

        //ちょっとあいまい
        //  dE           dE                da(cross_entro)      
        // ---- =   -----------------  * ------------------
        //  da           dy                da(softmax)

        //クロスエントロピー誤差の微分は、 -t / y ; -t*log(y)
        //	{  y'  =  -1 * (t / (y + EPS));   }
            //さらに、ソフトマックスの微分
        //	{ y * (1 - y);						}
//        printf("dEdy softmax[%d] => %lf  (y:%lf  t:%lf) \n", _idx, y - t ,y ,t );
        return y - t;       //※yに、ソフトマックスの結果を入れる。
//        return softmax_y - t;
}
*/



//ソフトマックスは
//
//              exp(an - amax)
//      ----------------------------------------
//          Σ   exp( ai  - amax)
//          i
//
template<size_t N>
__device__ void net<N>::loss_softmax_with_crossentropy()
{
    layer &l  = layers[N-1];    //最終レイヤー
    //GPUを利用して、MAXを計算します。
   int x = threadIdx.x;


   //一番近い2のs累乗の数字を探す。
   {
       int _n = 1;
        while (_n < N) {
            _n <<= 1;  // 2倍していく
        }
        //ここで
   }

   float max[N];    //
   //奇数だったらどうなるか
   for(int n=N/2 ; n>1 ; n/=2 ){                                //N=7 : n=3         n=1
        if(x < n){                                              //thread.xは0,1,2 
            max[x] = l.nondes.gpu[x] > l.nodes.gpu[x+n];        //max(a[0],a[3])   , max(a[1],a[4]) , max[[2] ,[5] ,   
                                                                //      max[0]          max[1]          max[2]
                                                                // 
        }
        __syncthreads();
   }
   //ここでmax[0]が最大値になっています。
   //奇数の場合、max[0]
   if(N%2!=0){

   }




}



#if 0
template<size_t N>
__device__ void net<N>::loss_softmax_with_crossentropy()
{
    //最終レイヤーに対して処理を施します。

    //gpu上での
    int node = threadIdx.x;
    //最終レイヤーのノードに対してのみ行う。
    if(node > layers[n_layer].size){        //
        goto _calc_dEda;
    }

    float cross_entropy_loss;           //損失関数です。
    {
        //    double  softmax_y;

        float softmax;

        //ソフトマックスを計算するときは、最終レイヤの 最大値と、合計を計算する必要がある。


        
    	float sum = 0.0;
        {
            va_list ap;	va_start( ap , t );						//第一引数に perceptron*
            const perceptron*p = va_arg(ap,const perceptron*);
    	    va_end(ap);
            double max = -100000.0;	//マイナスの大きな値にしておきます。
		    //まずmaxを計算します。
		    //
		    //tokuこれがだめ。pは、自分のノード。
		    //同じ列の入力を見ないといけないので、in_array()のout()を見る必要があります。
		    //affineの後なので、本来、同レイヤーの
    		for (int i = 0; i < p->in_array()->size(); ++i ) {		if(p->in_array()->at(i)->out() > max)		{ max = p->in_array()->at(i)->out(); }	}	//maxを算出していきます。
	    	for (int i = 0; i < p->in_array()->size(); ++i ) {
		    	sum += exp( p->in_array()->at(i)->out() - max);
		    }	//これはaffin後の入力です。
		    //今回の活性化の値は、以下です。
            //		_Assert(sum!=0.0 , "softmax::act() : sum is zero");
		    if(sum==0.0)sum=0.00000001;	//0だった場合にはとても小さな値にしてみる。
		    softmax_y = 	exp(p->a()-max) / sum;//ソフトマックスと、クロスエントロピー誤差を組み合わせた損失関数
    	    //^^^^^^
        }
        cross_entropy_y = -1.0 * t * log(softmax_y);
    }
_calc_dEda:
    ;

}
#endif